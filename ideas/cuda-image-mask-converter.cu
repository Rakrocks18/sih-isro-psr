#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <opencv2/opencv.hpp>

// CUDA kernel for applying mask and converting image
__global__ void applyMaskKernel(const uchar4* inputImage, const unsigned char* mask, 
                                uchar4* outputImage, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        int idx = y * width + x;
        uchar4 pixel = inputImage[idx];
        unsigned char maskValue = mask[idx];
        
        // Apply mask: if mask > 128, keep original alpha, else set to 0
        pixel.w = (maskValue > 128) ? 255 : 0;
        
        outputImage[idx] = pixel;
    }
}

// Host function to set up and launch the CUDA kernel
void applyMaskCUDA(const cv::Mat& inputImage, const cv::Mat& maskImage, cv::Mat& outputImage) {
    int width = inputImage.cols;
    int height = inputImage.rows;
    
    // Allocate device memory
    uchar4* d_inputImage;
    unsigned char* d_mask;
    uchar4* d_outputImage;
    
    hipMalloc(&d_inputImage, width * height * sizeof(uchar4));
    hipMalloc(&d_mask, width * height * sizeof(unsigned char));
    hipMalloc(&d_outputImage, width * height * sizeof(uchar4));
    
    // Copy input data to device
    hipMemcpy(d_inputImage, inputImage.ptr<uchar4>(), width * height * sizeof(uchar4), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, maskImage.ptr<unsigned char>(), width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Set up grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    
    // Launch kernel
    applyMaskKernel<<<gridSize, blockSize>>>(d_inputImage, d_mask, d_outputImage, width, height);
    
    // Copy result back to host
    hipMemcpy(outputImage.ptr<uchar4>(), d_outputImage, width * height * sizeof(uchar4), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_inputImage);
    hipFree(d_mask);
    hipFree(d_outputImage);
}

int main() {
    // Load input image and mask
    cv::Mat inputImage = cv::imread("path/to/input.jpg", cv::IMREAD_UNCHANGED);
    cv::Mat maskImage = cv::imread("path/to/mask.png", cv::IMREAD_GRAYSCALE);
    
    if (inputImage.empty() || maskImage.empty()) {
        printf("Error: Could not read input images\n");
        return -1;
    }
    
    // Ensure the dimensions match
    if (inputImage.size() != maskImage.size()) {
        printf("Error: Input image and mask dimensions do not match\n");
        return -1;
    }
    
    // Create output image
    cv::Mat outputImage(inputImage.size(), CV_8UC4);
    
    // Apply mask using CUDA
    applyMaskCUDA(inputImage, maskImage, outputImage);
    
    // Save output image
    cv::imwrite("path/to/output.png", outputImage);
    
    printf("Converted image saved to path/to/output.png\n");
    
    return 0;
}
